#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <stdbool.h>
#include <hipblas.h>
#include "1.cuh"
int main()
{
    hipblasHandle_t s222;
    hipblasCreate(&s222);
    float al = 1, ve = 0;
    float a[5 * 10] = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
    float b[5 * 5] = {2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0};
    float *d_a;
    float *d_b;
    float *d_c;
    hipMalloc(&d_a, 50 * sizeof(float));
    hipMalloc(&d_b, 25 * sizeof(float));
    hipMalloc(&d_c, 50 * sizeof(float));
    hipMemcpy(d_a, a, 50 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, 25 * sizeof(float), hipMemcpyHostToDevice);

    hipblasSgemm(s222,
                   HIPBLAS_OP_N, HIPBLAS_OP_N,
                   10, 5, 5,
                   &al,
                   d_a, 10,
                   d_b, 5,
                   &ve,
                   d_c, 10);
    hipDeviceSynchronize();

    float p[50];
    hipMemcpy(p, d_c, 50 * sizeof(float), hipMemcpyDeviceToHost);

    double time;
    // gemm_cublas(a, b, p, 10, 5, 5, &time);
    for (int sd = 0; sd < 50; sd++)
    {
        printf("%f ", p[sd]);
    }
    return 0;
}
