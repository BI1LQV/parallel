#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <stdbool.h>
#include "mmio.h"
#include "mmiohighlevel.h"
#include <hipblas.h>
#include <openblas_config.h>
#include <generated/cblas.h>
#include <omp.h>

#include <string.h>
#include <unistd.h>
#include <arpa/inet.h>
#include <sys/socket.h>
#include <hipsparse.h>
#define cycleTime 120
#define SPLIT_BLOCK 100
#define SPLIT_THREAD 256
#define CPU_SPLIT 60000
#define BIAS -0.3
#define BATCH_SIZE 60000
typedef struct
{
	VALUE_TYPE *value;
	int *columnindex;
	int *rowpointer;

} SMatrix;
void toColIndx_(int line, int ld, VALUE_TYPE *val)
{
	VALUE_TYPE *temp = (VALUE_TYPE *)malloc(sizeof(VALUE_TYPE) * line * ld);

	for (int i = 0; i < ld; ++i)
	{
		for (int j = 0; j < line; ++j)
		{
			temp[i * line + j] = val[j * ld + i];
		}
	}
	memcpy(val, temp, sizeof(VALUE_TYPE) * line * ld);
	free(temp);
}

void toRowIndx_(int line, int ld, VALUE_TYPE *val)
{
	VALUE_TYPE *temp = (VALUE_TYPE *)malloc(sizeof(VALUE_TYPE) * line * ld);

	for (int i = 0; i < line; ++i)
	{
		for (int j = 0; j < ld; ++j)
		{
			temp[i * ld + j] = val[j * line + i];
		}
	}
	memcpy(val, temp, sizeof(VALUE_TYPE) * line * ld);
	free(temp);
}
__global__ void relu(VALUE_TYPE *d_C0_value, int mC, int nC)
{
	int i = (blockIdx.x * SPLIT_BLOCK + blockIdx.y) * blockDim.x * blockDim.y + threadIdx.x * SPLIT_THREAD + threadIdx.y;
	VALUE_TYPE tmp = BIAS;
	tmp += d_C0_value[i];
	if (tmp <= 0)
	{
		tmp = 0;
	}
	else if (tmp >= 32)
	{
		tmp = 32;
	}
	d_C0_value[i] = tmp;
}
void calc(timeval t1, timeval t2,
		  VALUE_TYPE *d_C0_value, int mC, int nC,
		  VALUE_TYPE *d_A0_dense_value, VALUE_TYPE **d_B_value, int mB, int cycleTime_var,
		  VALUE_TYPE **B_csc_value, int **B_csc_rowIdx, int **B_csc_colPtr, int *b_csc_nnz)
{
	VALUE_TYPE al = 1, ve = 0;
	dim3 dimGrid(mC / SPLIT_BLOCK, SPLIT_BLOCK);
	dim3 dimBlock(nC / SPLIT_THREAD, SPLIT_THREAD);
	for (int k = 0; k < cycleTime_var; k++)
	{
		gettimeofday(&t1, NULL);
		// calc c=a*b
		// hipblasHandle_t s;
		// hipblasCreate(&s);

		// hipblasSgemm(s,
		// 			   HIPBLAS_OP_N, HIPBLAS_OP_N,
		// 			   BATCH_SIZE, 1024, 1024,
		// 			   &al,
		// 			   d_A0_dense_value, BATCH_SIZE,
		// 			   d_B_value[k], mB,
		// 			   &ve,
		// 			   d_C0_value, BATCH_SIZE);
		hipsparseHandle_t handle;
		hipsparseCreate(&handle);
		float a = 1;
		float b = 0;

		hipsparseSgemmi(handle,
					   BATCH_SIZE,
					   1024,
					   1024,
					   b_csc_nnz[k],
					   &a,
					   d_A0_dense_value,
					   BATCH_SIZE,
					   B_csc_value[k],
					   B_csc_colPtr[k],
					   B_csc_rowIdx[k],
					   &b,
					   d_C0_value,
					   BATCH_SIZE);
		hipDeviceSynchronize();

		gettimeofday(&t2, NULL);
		double time_gemm = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;

		gettimeofday(&t1, NULL);

		relu<<<dimGrid, dimBlock>>>(d_C0_value, mC, nC);
		hipDeviceSynchronize();
		gettimeofday(&t2, NULL);
		double time_biasrelu = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
		printf("k = %d, GEMM time: %4.5f ms, Bias+ReLU time: %4.5f ms\n",
			   k + 1, time_gemm, time_biasrelu);

		hipMemcpy(d_A0_dense_value, d_C0_value, (BATCH_SIZE * nC) * sizeof(VALUE_TYPE), hipMemcpyDeviceToDevice);
	}
}

int main(int argc, char **argv)
{
	struct timeval t1, t2, t3, t4;
	int size1 = 0;
	int size2 = 0;
	int *tc1;
	int *tc2;

	int mA;
	int nA;
	int nnzA;
	int isSymmetricA;
	SMatrix A;

	int mB;
	int nB;
	int nnzB;
	int isSymmetricB;
	SMatrix B[120];

	int mC, nC;
	int nnzC_golden = 0;

	// int sock = socket(AF_INET, SOCK_STREAM, 0);
	// struct sockaddr_in serv_addr;
	// memset(&serv_addr, 0, sizeof(serv_addr));
	// serv_addr.sin_family = AF_INET;
	// serv_addr.sin_addr.s_addr = inet_addr("127.0.0.1");
	// serv_addr.sin_port = htons(1234);
	// connect(sock, (struct sockaddr *)&serv_addr, sizeof(serv_addr));

	// load A data from file
	gettimeofday(&t3, NULL);
	char filename1[] = "sparse-images-1024.tsv";
	mmio_info(&mA, &nA, &nnzA, &isSymmetricA, filename1);
	A.value = (VALUE_TYPE *)malloc((nnzA) * sizeof(VALUE_TYPE));
	A.columnindex = (int *)malloc((nnzA) * sizeof(int));
	A.rowpointer = (int *)malloc((mA + 1) * sizeof(int));
	mmio_data(A.rowpointer, A.columnindex, A.value, filename1);
	printf("input matrix A: ( %i, %i ) nnz = %i\n", mA, nA, nnzA);
	VALUE_TYPE *A0_dense_value = (VALUE_TYPE *)malloc(mA * nA * sizeof(VALUE_TYPE));

	memset(A0_dense_value, 0, sizeof(VALUE_TYPE) * mA * nA);
	for (int i = 0; i < mA; i++)
	{
		for (int j = A.rowpointer[i]; j < A.rowpointer[i + 1]; j++)
		{
			A0_dense_value[i * nA + A.columnindex[j]] = A.value[j];
		}
	}
	VALUE_TYPE *d_A0_dense_split_value[60000 / BATCH_SIZE];
	VALUE_TYPE *d_C0_split_value[60000 / BATCH_SIZE];
	VALUE_TYPE *tmp = (VALUE_TYPE *)malloc(BATCH_SIZE * nA * sizeof(VALUE_TYPE));
	for (int splice = (60000 / BATCH_SIZE) - 1; splice >= 0; splice--)
	{
		hipMalloc(&d_A0_dense_split_value[splice], (BATCH_SIZE * nA) * sizeof(VALUE_TYPE));
		hipMalloc(&d_C0_split_value[splice], (BATCH_SIZE * nA) * sizeof(VALUE_TYPE));
		hipMemcpy(tmp, A0_dense_value + splice * (BATCH_SIZE * nA), (BATCH_SIZE * nA) * sizeof(VALUE_TYPE), hipMemcpyHostToHost);
		toColIndx_(BATCH_SIZE, 1024, tmp);
		hipMemcpy(d_A0_dense_split_value[splice], tmp, (BATCH_SIZE * nA) * sizeof(VALUE_TYPE), hipMemcpyHostToDevice);
	}

	char neuronfile1[] = "neuron1024/n1024-l";
	char neuronfile2[] = ".tsv";
	char filename3[60];

	VALUE_TYPE *d_B_value[120];
	VALUE_TYPE *B_value[120];
	VALUE_TYPE *B_csc_value[120];
	int *B_csc_rowIdx[120];
	int *B_csc_colPtr[120];
	int *B_csc_nnz = (int *)malloc(120 * sizeof(int));

	for (int k = 0; k < cycleTime; k++)
	{
		char filenum[5];
		int k1 = k + 1;
		snprintf(filenum, sizeof(filenum), "%d", k1);

		strcpy(filename3, neuronfile1);
		strcat(filename3, filenum);
		strcat(filename3, neuronfile2);

		mmio_info(&mB, &nB, &nnzB, &isSymmetricB, filename3);
		B_csc_nnz[k] = nnzB;
		B[k].value = (VALUE_TYPE *)malloc((nnzB) * sizeof(VALUE_TYPE));
		B[k].columnindex = (int *)malloc((nnzB) * sizeof(int));
		B[k].rowpointer = (int *)malloc((mB + 1) * sizeof(int));
		mmio_data(B[k].rowpointer, B[k].columnindex, B[k].value, filename3);

		B_value[k] = (VALUE_TYPE *)malloc(mB * nB * sizeof(VALUE_TYPE));

		memset(B_value[k], 0, sizeof(VALUE_TYPE) * mB * nB);
		for (int i = 0; i < mB; i++)
		{
			for (int j = B[k].rowpointer[i]; j < B[k].rowpointer[i + 1]; j++)
			{
				B_value[k][i * nB + B[k].columnindex[j]] = B[k].value[j];
			}
		}
		hipMalloc(&d_B_value[k], sizeof(VALUE_TYPE) * mB * nB);
		hipMemcpy(d_B_value[k], B_value[k], sizeof(VALUE_TYPE) * mB * nB,
				   hipMemcpyHostToDevice);

		hipMalloc(&B_csc_value[k], (nnzB) * sizeof(VALUE_TYPE));
		hipMalloc(&B_csc_rowIdx[k], (nnzB) * sizeof(int));
		hipMalloc(&B_csc_colPtr[k], (mB + 1) * sizeof(int));

		int dataNumInCol = 0;
		int B_csc_value_idx = 0;
		int B_csc_rowIdx_idx = 0;
		int B_csc_colPtr_idx = 0;
		float *B_csc_value_tmp = (VALUE_TYPE *)malloc((nnzB) * sizeof(VALUE_TYPE));
		int *B_csc_rowIdx_tmp = (int *)malloc((nnzB) * sizeof(int));
		int *B_csc_colPtr_tmp = (int *)malloc((mB + 1) * sizeof(int));
		B_csc_colPtr_tmp[0] = 0;
		for (int colIdx = 0; colIdx < 1024; colIdx++)
		{
			for (int rowIdx = 0; rowIdx < 1024; rowIdx++)
			{
				if (B_value[k][rowIdx * 1024 + colIdx])
				{
					B_csc_value_tmp[B_csc_value_idx++] = B_value[k][rowIdx * 1024 + colIdx];
					dataNumInCol++;
					B_csc_rowIdx_tmp[B_csc_rowIdx_idx++] = rowIdx;
				}
			}
			B_csc_colPtr_tmp[B_csc_colPtr_idx + 1] = B_csc_colPtr_tmp[B_csc_colPtr_idx] + dataNumInCol;
			B_csc_colPtr_idx++;
			dataNumInCol = 0;
		}
		hipMemcpy(B_csc_value[k], B_csc_value_tmp, (nnzB) * sizeof(VALUE_TYPE), hipMemcpyHostToDevice);
		hipMemcpy(B_csc_rowIdx[k], B_csc_rowIdx_tmp, (nnzB) * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(B_csc_colPtr[k], B_csc_colPtr_tmp, (mB + 1) * sizeof(int), hipMemcpyHostToDevice);

		for (int x = 0; x < mB; x++)
		{
			for (int y = 0; y < x; y++)
			{
				VALUE_TYPE tmp;
				tmp = B_value[k][y * mB + x];
				B_value[k][y * mB + x] = B_value[k][x * mB + y];
				B_value[k][x * mB + y] = tmp;
			}
		}
	}

	mC = BATCH_SIZE;
	nC = nB;
	// warm up
	printf("---------warm up------------\n");
	calc(t1, t2,
		 d_C0_split_value[0], mC, nC,
		 d_A0_dense_split_value[0], d_B_value, mB, 5,
		 B_csc_value, B_csc_rowIdx, B_csc_colPtr, B_csc_nnz);
	//清空d_a0
	hipMemcpy(d_A0_dense_split_value[0], tmp, BATCH_SIZE * nA * sizeof(VALUE_TYPE),
			   hipMemcpyHostToDevice);
	printf("---------warm up end------------\n");
	gettimeofday(&t4, NULL);
	double time_load = (t4.tv_sec - t3.tv_sec) * 1000.0 + (t4.tv_usec - t3.tv_usec) / 1000.0;
	printf("Weight matrix load and warm up time: %f ms \n", time_load);

	gettimeofday(&t3, NULL);
	for (int st = 0; st < 60000 / BATCH_SIZE; st++)
	{
		calc(t1, t2,
			 d_C0_split_value[st], mC, nC,
			 d_A0_dense_split_value[st], d_B_value, mB, cycleTime,
			 B_csc_value, B_csc_rowIdx, B_csc_colPtr, B_csc_nnz);
	}

	gettimeofday(&t4, NULL);
	double time_inference = (t4.tv_sec - t3.tv_sec) * 1000.0 + (t4.tv_usec - t3.tv_usec) / 1000.0;
	printf("Inference time: %f ms \n", time_inference);

	VALUE_TYPE *A0 = (VALUE_TYPE *)malloc(CPU_SPLIT * 1024 * sizeof(VALUE_TYPE));
	VALUE_TYPE *tmp2 = (VALUE_TYPE *)malloc(BATCH_SIZE * 1024 * sizeof(VALUE_TYPE));
	for (int splice = 0; splice < 60000 / BATCH_SIZE; splice++)
	{
		hipMemcpy(tmp2, d_A0_dense_split_value[splice], BATCH_SIZE * 1024 * sizeof(VALUE_TYPE), hipMemcpyDeviceToHost);
		toRowIndx_(BATCH_SIZE, 1024, tmp2);
		hipMemcpy(A0 + splice * BATCH_SIZE * 1024, tmp2, BATCH_SIZE * 1024 * sizeof(VALUE_TYPE), hipMemcpyHostToHost);
	}

	//  check results
	// TODO: upload
	printf("test\n");
	FILE *fs;
	fs = fopen("sparse-images-1024-1.tsv", "w+");
	int i = 0;
	for (; i < CPU_SPLIT; i++)
	{
		int sum = 0;
		for (int j = (i * nA); j < ((i + 1) * nA); j++)
		{
			sum += A0[j];
		}

		if (sum != 0)
		{
			fprintf(fs, "%d\n", i + 1);
		}
	}

	fclose(fs);
	FILE *fp2 = NULL;

	fp2 = fopen("sparse-images-1024-1.tsv", "rb");
	if (fp2 == NULL)
	{
		printf("Error:Open file fail!\n");
	}

	fseek(fp2, 0, SEEK_END);
	size2 = ftell(fp2);
	rewind(fp2);

	tc2 = (int *)malloc(sizeof(int) * size2 / 4);

	int readnum2 = fread(tc2, 4, size2 / 4, fp2);

	fclose(fp2);

	FILE *fp1;

	fp1 = fopen("neuron1024-l120-categories.tsv", "rb");
	if (fp1 == NULL)
	{
		printf("Error:Open file fail!\n");
	}

	fseek(fp1, 0, SEEK_END);
	size1 = ftell(fp1);
	rewind(fp1);

	tc1 = (int *)malloc(sizeof(int) * size1 / 4);

	int readnum1 = fread(tc1, 4, size1 / 4, fp1);

	fclose(fp1);
	int judge = 0;
	for (int i = 0; i < size1 / 4; i++)
	{
		if (tc1[i] - tc2[i] != 0)
		{
			judge++;
		}
	}
	printf("judge:%d\n", judge);
	if (judge == 0)
	{
		printf("CHALLENGE PASSED\n");
	}
	else
	{
		printf("CHALLENGE FAILED\n");
	}

	free(A0);

	return 0;
}
