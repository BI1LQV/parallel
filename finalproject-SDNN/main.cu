#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <stdbool.h>
#include "mmio.h"
#include "mmiohighlevel.h"
#include <hipblas.h>
#include <openblas_config.h>
#include <generated/cblas.h>
#include <omp.h>
#define cycleTime 120
#define SPLIT_BLOCK 100
#define SPLIT_THREAD 256
#define CPU_SPLIT 59000
typedef struct
{
	VALUE_TYPE *value;
	int *columnindex;
	int *rowpointer;

} SMatrix;
void toColIndx_(int line, int ld, VALUE_TYPE *val)
{
	VALUE_TYPE *temp = (VALUE_TYPE *)malloc(sizeof(VALUE_TYPE) * line * ld);

	for (int i = 0; i < ld; ++i)
	{
		for (int j = 0; j < line; ++j)
		{
			temp[i * line + j] = val[j * ld + i];
		}
	}
	memcpy(val, temp, sizeof(VALUE_TYPE) * line * ld);
	free(temp);
}

void toRowIndx_(int line, int ld, VALUE_TYPE *val)
{
	VALUE_TYPE *temp = (VALUE_TYPE *)malloc(sizeof(VALUE_TYPE) * line * ld);

	for (int i = 0; i < line; ++i)
	{
		for (int j = 0; j < ld; ++j)
		{
			temp[i * ld + j] = val[j * line + i];
		}
	}
	memcpy(val, temp, sizeof(VALUE_TYPE) * line * ld);
	free(temp);
}
__global__ void relu(VALUE_TYPE *d_C0_value, int mC, int nC)
{
	int i = (blockIdx.x * SPLIT_BLOCK + blockIdx.y) * blockDim.x * blockDim.y + threadIdx.x * SPLIT_THREAD + threadIdx.y;
	VALUE_TYPE tmp = -0.3;
	tmp += d_C0_value[i];
	if (tmp <= 0)
	{
		tmp = 0;
	}
	else if (tmp >= 32)
	{
		tmp = 32;
	}
	d_C0_value[i] = tmp;
}
void calc(timeval t1, timeval t2,
		  VALUE_TYPE *d_C0_value, int mC, int nC, int mA,
		  VALUE_TYPE *d_A0_dense_value, VALUE_TYPE **d_B_value, int mB, int cycleTime_var)
{
	for (int k = 0; k < cycleTime_var; k++)
	{
		gettimeofday(&t1, NULL);
		// calc c=a*b
		hipblasHandle_t s;
		hipblasCreate(&s);
		VALUE_TYPE al = 1, ve = 0;

		hipblasSgemm(s,
					   HIPBLAS_OP_N, HIPBLAS_OP_N,
					   mA, 1024, 1024,
					   &al,
					   d_A0_dense_value, mA,
					   d_B_value[k], mB,
					   &ve,
					   d_C0_value, mA);
		hipDeviceSynchronize();

		gettimeofday(&t2, NULL);
		double time_gemm = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;

		gettimeofday(&t1, NULL);
		dim3 dimGrid(mC / SPLIT_BLOCK, SPLIT_BLOCK);
		dim3 dimBlock(nC / SPLIT_THREAD, SPLIT_THREAD);
		relu<<<dimGrid, dimBlock>>>(d_C0_value, mC, nC);
		hipDeviceSynchronize();
		gettimeofday(&t2, NULL);
		double time_biasrelu = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
		printf("k = %d, GEMM time: %4.5f ms, Bias+ReLU time: %4.5f ms\n",
			   k + 1, time_gemm, time_biasrelu);

		hipMemcpy(d_A0_dense_value, d_C0_value, (mC * nC) * sizeof(VALUE_TYPE), hipMemcpyDeviceToDevice);
	}
}

void calcByCPU(VALUE_TYPE *A_dense, VALUE_TYPE **B_dense, VALUE_TYPE *C_dense, int lengthA)
{
	struct timeval tq, tr;
	enum CBLAS_ORDER order = CblasRowMajor;
	enum CBLAS_TRANSPOSE transposeA = CblasNoTrans;
	enum CBLAS_TRANSPOSE transposeB = CblasNoTrans;
	double alpha = 1;
	double beta = 0;
	int m = lengthA;
	int n = 1024;
	int k = 1024;
	double time_gemm, time_biasrelu;
	for (int kp = 0; kp < cycleTime; kp++)
	{
		gettimeofday(&tq, NULL);
		cblas_sgemm(order, transposeA, transposeB, m, n, k, alpha, A_dense, k, B_dense[kp], n, beta, C_dense, n);
		gettimeofday(&tr, NULL);
		time_gemm = (tr.tv_sec - tq.tv_sec) * 1000.0 + (tr.tv_usec - tq.tv_usec) / 1000.0;
		// #pragma omp parallel for
		for (int i = 0; i < lengthA * 1024; i++)
		{
			C_dense[i] += -0.3;
			if (C_dense[i] <= 0)
			{
				C_dense[i] = 0;
			}
			else if (C_dense[i] >= 32)
			{
				C_dense[i] = 32;
			}
		}
		gettimeofday(&tq, NULL);
		time_biasrelu = (tq.tv_sec - tr.tv_sec) * 1000.0 + (tq.tv_usec - tr.tv_usec) / 1000.0;
		printf("k = %d,CPU GEMM time: %4.5f ms, Bias+ReLU time: %4.5f ms\n",
			   kp + 1, time_gemm, time_biasrelu);
		hipMemcpy(A_dense, C_dense, (m * n) * sizeof(VALUE_TYPE), hipMemcpyHostToHost);
	}
}
int main(int argc, char **argv)
{
	struct timeval t1, t2, t3, t4;
	int size1 = 0;
	int size2 = 0;
	int *tc1;
	int *tc2;
	VALUE_TYPE bias = -0.3000;

	int mA;
	int nA;
	int nnzA;
	int isSymmetricA;
	SMatrix A;

	int mB;
	int nB;
	int nnzB;
	int isSymmetricB;
	SMatrix B[120];

	int mC, nC;
	int nnzC_golden = 0;

	// load A data from file
	gettimeofday(&t3, NULL);
	char filename1[] = "sparse-images-1024.tsv";
	mmio_info(&mA, &nA, &nnzA, &isSymmetricA, filename1);
	A.value = (VALUE_TYPE *)malloc((nnzA) * sizeof(VALUE_TYPE));
	A.columnindex = (int *)malloc((nnzA) * sizeof(int));
	A.rowpointer = (int *)malloc((mA + 1) * sizeof(int));
	mmio_data(A.rowpointer, A.columnindex, A.value, filename1);
	printf("input matrix A: ( %i, %i ) nnz = %i\n", mA, nA, nnzA);
	VALUE_TYPE *A0_dense_value = (VALUE_TYPE *)malloc(mA * nA * sizeof(VALUE_TYPE));
	VALUE_TYPE *d_A0_dense_value;
	hipMalloc(&d_A0_dense_value, mA * nA * sizeof(VALUE_TYPE));
	memset(A0_dense_value, 0, sizeof(VALUE_TYPE) * mA * nA);
	for (int i = 0; i < mA; i++)
	{
		for (int j = A.rowpointer[i]; j < A.rowpointer[i + 1]; j++)
		{
			A0_dense_value[i * nA + A.columnindex[j]] = A.value[j];
		}
	}
	mA = CPU_SPLIT;
	VALUE_TYPE *A_dense_cpu = (VALUE_TYPE *)malloc((60000 - mA) * nA * sizeof(VALUE_TYPE));
	hipMemcpy(A_dense_cpu, A0_dense_value + mA * nA, (60000 - mA) * nA * sizeof(VALUE_TYPE), hipMemcpyHostToHost);
	VALUE_TYPE *C_dense_cpu = (VALUE_TYPE *)malloc((60000 - mA) * nA * sizeof(VALUE_TYPE));
	memset(C_dense_cpu, 0, sizeof(VALUE_TYPE) * (60000 - mA) * nA);
	toColIndx_(mA, 1024, A0_dense_value);

	hipMemcpy(d_A0_dense_value, A0_dense_value, mA * nA * sizeof(VALUE_TYPE),
			   hipMemcpyHostToDevice);

	char neuronfile1[] = "neuron1024/n1024-l";
	char neuronfile2[] = ".tsv";
	char filename3[60];

	VALUE_TYPE *d_B_value[120];
	VALUE_TYPE *B_value[120];
	VALUE_TYPE *B_value_cpu[120];
	for (int k = 0; k < cycleTime; k++)
	{
		char filenum[5];
		int k1 = k + 1;
		snprintf(filenum, sizeof(filenum), "%d", k1);

		strcpy(filename3, neuronfile1);
		strcat(filename3, filenum);
		strcat(filename3, neuronfile2);

		mmio_info(&mB, &nB, &nnzB, &isSymmetricB, filename3);
		B[k].value = (VALUE_TYPE *)malloc((nnzB) * sizeof(VALUE_TYPE));
		B[k].columnindex = (int *)malloc((nnzB) * sizeof(int));
		B[k].rowpointer = (int *)malloc((mB + 1) * sizeof(int));
		mmio_data(B[k].rowpointer, B[k].columnindex, B[k].value, filename3);

		B_value[k] = (VALUE_TYPE *)malloc(mB * nB * sizeof(VALUE_TYPE));
		B_value_cpu[k] = (VALUE_TYPE *)malloc(mB * nB * sizeof(VALUE_TYPE));
		memset(B_value[k], 0, sizeof(VALUE_TYPE) * mB * nB);
		for (int i = 0; i < mB; i++)
		{
			for (int j = B[k].rowpointer[i]; j < B[k].rowpointer[i + 1]; j++)
			{
				B_value[k][i * nB + B[k].columnindex[j]] = B[k].value[j];
			}
		}
		hipMemcpy(B_value_cpu[k], B_value[k], sizeof(VALUE_TYPE) * mB * nB,
				   hipMemcpyHostToHost);
		for (int x = 0; x < mB; x++)
		{
			for (int y = 0; y < x; y++)
			{
				VALUE_TYPE tmp;
				tmp = B_value[k][y * mB + x];
				B_value[k][y * mB + x] = B_value[k][x * mB + y];
				B_value[k][x * mB + y] = tmp;
			}
		}

		hipMalloc(&d_B_value[k], sizeof(VALUE_TYPE) * mB * nB);
		hipMemcpy(d_B_value[k], B_value[k], sizeof(VALUE_TYPE) * mB * nB,
				   hipMemcpyHostToDevice);
	}

	VALUE_TYPE *d_C0_value;
	hipMalloc(&d_C0_value, (mA * nA) * sizeof(VALUE_TYPE));
	// warm up
	printf("---------warm up------------\n");
	calc(t1, t2,
		 d_C0_value, mC, nC, mA,
		 d_A0_dense_value, d_B_value, mB, 20);
	//清空d_a0
	hipMemcpy(d_A0_dense_value, A0_dense_value, mA * nA * sizeof(VALUE_TYPE),
			   hipMemcpyHostToDevice);
	printf("---------warm up end------------\n");
	gettimeofday(&t4, NULL);
	double time_load = (t4.tv_sec - t3.tv_sec) * 1000.0 + (t4.tv_usec - t3.tv_usec) / 1000.0;
	printf("Weight matrix load and warm up time: %f ms \n", time_load);

	mC = mA;
	nC = nB;
	gettimeofday(&t3, NULL);
#pragma omp parallel
	{
#pragma omp single
		{
#pragma omp task
			calc(t1, t2,
				 d_C0_value, mC, nC, mA,
				 d_A0_dense_value, d_B_value, mB, cycleTime);
#pragma omp task
			calcByCPU(A_dense_cpu, B_value_cpu, C_dense_cpu, 60000 - mA);
		}
	}

	// calc(t1, t2,
	// 	 d_C0_value, mC, nC, mA,
	// 	 d_A0_dense_value, d_B_value, mB, cycleTime);

	gettimeofday(&t4, NULL);
	double time_inference = (t4.tv_sec - t3.tv_sec) * 1000.0 + (t4.tv_usec - t3.tv_usec) / 1000.0;
	printf("Inference time: %f ms \n", time_inference);

	VALUE_TYPE *A0 = (VALUE_TYPE *)malloc(mA * 1024 * sizeof(VALUE_TYPE));
	hipMemcpy(A0, d_A0_dense_value, mA * 1024 * sizeof(VALUE_TYPE), hipMemcpyDeviceToHost);
	// 转置
	toRowIndx_(mA, 1024, A0);
	//  check results
	printf("test\n");
	FILE *fs;
	fs = fopen("sparse-images-1024-1.tsv", "w+");
	int i = 0;
	for (; i < mA; i++)
	{
		int sum = 0;
		for (int j = (i * nA); j < ((i + 1) * nA); j++)
		{
			sum += A0[j];
		}

		if (sum != 0)
		{
			fprintf(fs, "%d\n", i + 1);
		}
	}
	for (; i < 60000; i++)
	{
		int sum = 0;
		int _i = i - mA;
		for (int j = (_i * nA); j < ((_i + 1) * nA); j++)
		{
			sum += A_dense_cpu[j];
		}

		if (sum != 0)
		{
			fprintf(fs, "%d\n", i + 1);
		}
	}
	fclose(fs);
	FILE *fp2 = NULL;

	fp2 = fopen("sparse-images-1024-1.tsv", "rb");
	if (fp2 == NULL)
	{
		printf("Error:Open file fail!\n");
	}

	fseek(fp2, 0, SEEK_END);
	size2 = ftell(fp2);
	rewind(fp2);

	tc2 = (int *)malloc(sizeof(int) * size2 / 4);

	int readnum2 = fread(tc2, 4, size2 / 4, fp2);

	fclose(fp2);

	FILE *fp1;

	fp1 = fopen("neuron1024-l120-categories.tsv", "rb");
	if (fp1 == NULL)
	{
		printf("Error:Open file fail!\n");
	}

	fseek(fp1, 0, SEEK_END);
	size1 = ftell(fp1);
	rewind(fp1);

	tc1 = (int *)malloc(sizeof(int) * size1 / 4);

	int readnum1 = fread(tc1, 4, size1 / 4, fp1);

	fclose(fp1);
	int judge = 0;
	for (int i = 0; i < size1 / 4; i++)
	{
		if (tc1[i] - tc2[i] != 0)
		{
			judge++;
		}
	}
	printf("judge:%d\n", judge);
	if (judge == 0)
	{
		printf("CHALLENGE PASSED\n");
	}
	else
	{
		printf("CHALLENGE FAILED\n");
	}

	free(A0);

	return 0;
}
