#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <stdbool.h>
#include "mmio.h"
#include "mmiohighlevel.h"
#include <hipsparse.h>
#include <cstdio>
#include <cstdlib>
#include <cstring>
typedef struct
{
	VALUE_TYPE *value;
	int *columnindex;
	int *rowpointer;

} SMatrix;
__global__ void relu(VALUE_TYPE *d_C0_value, int mC, int nC)
{
	for (int i = 0; i < mC * nC; i++)
	{
		if (d_C0_value[i] <= 0)
		{
			d_C0_value[i] = 0;
		}
		else if (d_C0_value[i] >= 32)
		{
			d_C0_value[i] = 32;
		}
	}
}
int main(int argc, char **argv)
{
	struct timeval t1, t2, t3, t4;
	int size1 = 0;
	int size2 = 0;
	int *tc1;
	int *tc2;
	VALUE_TYPE bias = -0.3000;

	int mA;
	int nA;
	int nnzA;
	int isSymmetricA;
	SMatrix A;

	int mB;
	int nB;
	int nnzB;
	int isSymmetricB;
	SMatrix B[120];

	int mC, nC;
	int nnzC_golden = 0;

	// load A data from file
	gettimeofday(&t3, NULL);
	char filename1[] = "sparse-images-1024.tsv";
	mmio_info(&mA, &nA, &nnzA, &isSymmetricA, filename1);
	A.value = (VALUE_TYPE *)malloc((nnzA) * sizeof(VALUE_TYPE));
	A.columnindex = (int *)malloc((nnzA) * sizeof(int));
	A.rowpointer = (int *)malloc((mA + 1) * sizeof(int));
	mmio_data(A.rowpointer, A.columnindex, A.value, filename1);
	printf("input matrix A: ( %i, %i ) nnz = %i\n", mA, nA, nnzA);

	int *d_A_rowpointer, *d_A_columnindex;

	hipMalloc(&d_A_rowpointer, (mA + 1) * sizeof(int));
	hipMemcpy(d_A_rowpointer, A.rowpointer, (mA + 1) * sizeof(int),
			   hipMemcpyHostToDevice);

	hipMalloc(&d_A_columnindex, (60000 * 1024) * sizeof(int));
	hipMemcpy(d_A_columnindex, A.columnindex, (60000 * 1024) * sizeof(int),
			   hipMemcpyHostToDevice);

	float *d_A_value;
	hipMalloc(&d_A_value, (60000 * 1024) * sizeof(VALUE_TYPE));
	hipMemcpy(d_A_value, A.value, (60000 * 1024) * sizeof(VALUE_TYPE),
			   hipMemcpyHostToDevice);

	hipsparseSpMatDescr_t d_csr_A;
	hipsparseCreateCsr(&d_csr_A, (int64_t)mA, (int64_t)nA,
					  nnzA, d_A_rowpointer, d_A_columnindex, d_A_value,
					  HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
					  HIPSPARSE_INDEX_BASE_ZERO,
					  HIP_R_32F);

	VALUE_TYPE *A0_dense_value = (VALUE_TYPE *)malloc(mA * nA * sizeof(VALUE_TYPE));
	VALUE_TYPE *d_A0_dense_value;
	hipsparseDnMatDescr_t d_A0_dense_mat;

	memset(A0_dense_value, 0, sizeof(VALUE_TYPE) * mA * nA);
	for (int i = 0; i < mA; i++)
	{
		for (int j = A.rowpointer[i]; j < A.rowpointer[i + 1]; j++)
		{
			A0_dense_value[i * nA + A.columnindex[j]] = A.value[j];
		}
	}

	hipMemcpy(A0_dense_value, d_A0_dense_value, mA * nA * sizeof(VALUE_TYPE),
			   hipMemcpyDeviceToHost);

	hipsparseCreateDnMat(&d_A0_dense_mat, (int64_t)mA, (int64_t)nA,
						(int64_t)mA, d_A0_dense_value, HIP_R_32F, HIPSPARSE_ORDER_COL);

	char neuronfile1[] = "neuron1024/n1024-l";
	char neuronfile2[] = ".tsv";
	char filename3[60];

	hipsparseSpMatDescr_t B0[120];
	VALUE_TYPE *d_B_value[120];
	VALUE_TYPE *B_value[120];
	hipsparseDnMatDescr_t d_B_den_val[120];

	for (int k = 0; k < 120; k++)
	{
		char filenum[5];
		int k1 = k + 1;
		snprintf(filenum, sizeof(filenum), "%d", k1);

		strcpy(filename3, neuronfile1);
		strcat(filename3, filenum);
		strcat(filename3, neuronfile2);

		mmio_info(&mB, &nB, &nnzB, &isSymmetricB, filename3);
		B[k].value = (VALUE_TYPE *)malloc((nnzB) * sizeof(VALUE_TYPE));
		B[k].columnindex = (int *)malloc((nnzB) * sizeof(int));
		B[k].rowpointer = (int *)malloc((mB + 1) * sizeof(int));
		mmio_data(B[k].rowpointer, B[k].columnindex, B[k].value, filename3);

		int *d_Bk_rowpointer, *d_Bk_columnindex;

		hipMalloc(&d_Bk_rowpointer, (mB + 1) * sizeof(int));
		hipMemcpy(d_Bk_rowpointer, B[k].rowpointer, (mB + 1) * sizeof(int),
				   hipMemcpyHostToDevice);

		hipMalloc(&d_Bk_columnindex, (nnzB) * sizeof(int));
		hipMemcpy(d_Bk_columnindex, B[k].columnindex, (nnzB) * sizeof(int),
				   hipMemcpyHostToDevice);

		VALUE_TYPE *d_Bk_value;
		hipMalloc(&d_Bk_value, (nnzB) * sizeof(VALUE_TYPE));
		hipMemcpy(d_Bk_value, B[k].value, (nnzB) * sizeof(VALUE_TYPE),
				   hipMemcpyHostToDevice);

		hipsparseCreateCsr(&B0[k], (int64_t)mB, (int64_t)nB,
						  nnzB, d_Bk_rowpointer, d_Bk_columnindex, d_Bk_value,
						  HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
						  HIPSPARSE_INDEX_BASE_ZERO,
						  HIP_R_32F);

		B_value[k] = (VALUE_TYPE *)malloc(mB * nB * sizeof(VALUE_TYPE));
		memset(B_value[k], 0, sizeof(VALUE_TYPE) * mB * nB);
		for (int i = 0; i < mB; i++)
		{
			for (int j = B[k].rowpointer[i]; j < B[k].rowpointer[i + 1]; j++)
			{
				B_value[k][i * nB + B[k].columnindex[j]] = B[k].value[j];
			}
		}

		hipMemcpy(d_B_value[k], B_value[k], sizeof(VALUE_TYPE) * mB * nB,
				   hipMemcpyHostToDevice);

		hipsparseCreateDnMat(&d_B_den_val[k], (int64_t)mB, (int64_t)nB,
							(int64_t)mB, d_B_value[k], HIP_R_32F, HIPSPARSE_ORDER_COL);
		hipDeviceSynchronize();
	}
	gettimeofday(&t4, NULL);
	double time_load = (t4.tv_sec - t3.tv_sec) * 1000.0 + (t4.tv_usec - t3.tv_usec) / 1000.0;
	printf("Weight matrix load time: %f ms \n", time_load);

	mC = mA;
	nC = nB;

	VALUE_TYPE *d_C0_value, *C0_value;
	hipsparseDnMatDescr_t d_C0_den;
	hipMalloc(&d_C0_value, (60000 * 1024) * sizeof(VALUE_TYPE));
	hipsparseCreateDnMat(&d_C0_den, (int64_t)60000, (int64_t)1024,
						(int64_t)60000, d_C0_value, HIP_R_32F, HIPSPARSE_ORDER_COL);

	gettimeofday(&t3, NULL);
	for (int k = 0; k < 120; k++)
	{
		gettimeofday(&t1, NULL);
		hipMemset(d_C0_value, bias, sizeof(VALUE_TYPE) * mC * nC);
		hipsparseHandle_t handle;
		hipsparseCreate(&handle);
		// TODO: convert dense a to csr a

		int *nnzPerRowColumn, *nnzTotalDevHostPtr;
		hipsparseMatDescr_t descrA;
		descrA.hipsparseIndexBase_t = HIPSPARSE_INDEX_BASE_ZERO;
		descrA.hipsparseMatrixType_t = HIPSPARSE_MATRIX_TYPE_GENERAL;
		descrA.hipsparseDiagType_t = HIPSPARSE_DIAG_TYPE_NON_UNIT;
		descrA.hipsparseFillMode_t = HIPSPARSE_FILL_MODE_LOWER;
		hipsparseSnnz(handle,
					 HIPSPARSE_DIRECTION_COLUMN,
					 60000,
					 1024,
					 descrA,
					 d_A0_dense_value,
					 60000,
					 nnzPerRowColumn,
					 nnzTotalDevHostPtr);
		hipsparseSdense2csr(handle,
						   60000,
						   1024,
						   descrA,
						   d_A0_dense_value,
						   60000,
						   nnzPerRowColumn,
						   d_A_value,
						   d_A_rowpointer,
						   d_A_columnindex);

		hipsparseOperation_t Ap = HIPSPARSE_OPERATION_NON_TRANSPOSE;
		hipsparseOperation_t Bp = HIPSPARSE_OPERATION_NON_TRANSPOSE;
		VALUE_TYPE al = 1, be = 0;

		hipsparseSpMM(handle, Ap, Bp, &al, d_csr_A, d_B_den_val[k], &be, d_C0_den,
					 HIP_R_32F, HIPSPARSE_MM_ALG_DEFAULT, NULL);
		hipDeviceSynchronize();

		gettimeofday(&t2, NULL);
		double time_gemm = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;

		gettimeofday(&t1, NULL);
		relu<<<1, 1>>>(d_C0_value, mC, nC);
		hipDeviceSynchronize();
		gettimeofday(&t2, NULL);
		double time_biasrelu = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
		printf("k = %d, GEMM time: %4.5f ms, Bias+ReLU time: %4.5f ms\n",
			   k + 1, time_gemm, time_biasrelu);

		hipMemcpy(d_A0_dense_value, d_C0_value, (mC * nC) * sizeof(VALUE_TYPE), hipMemcpyDeviceToDevice);
	}

	gettimeofday(&t4, NULL);
	double time_inference = (t4.tv_sec - t3.tv_sec) * 1000.0 + (t4.tv_usec - t3.tv_usec) / 1000.0;
	printf("Inference time: %f ms \n", time_inference);

	// // check results
	// printf("test\n");
	// FILE* fs;
	// fs=fopen("sparse-images-1024-1.tsv","w+");
	// for (int i = 0; i <mA; i++)
	// {
	// 	int sum =0;
	// 	for (int j = (i*nA); j < ((i+1)*nA); j++)
	// 	{
	// 		sum+=A0[j];

	// 	}
	// 	if(sum!=0)
	// 	{
	// 		fprintf(fs,"%d\n", i+1);
	// 	}
	// }
	// fclose(fs);
	// FILE* fp2=NULL;

	// fp2 = fopen("sparse-images-1024-1.tsv", "rb");
	// if (fp2 == NULL)
	// {
	// 	printf("Error:Open file fail!\n");
	// }

	// fseek(fp2, 0, SEEK_END);
	// size2 = ftell(fp2);
	// rewind(fp2);

	// tc2 = (int*)malloc(sizeof(int) * size2/4);

	// int readnum2 = fread(tc2, 4, size2/4, fp2);

	// fclose(fp2);

	// FILE* fp1;

	// fp1 = fopen("neuron1024-l120-categories.tsv", "rb");
	// if (fp1 == NULL)
	// {
	// 	printf("Error:Open file fail!\n");
	// }

	// fseek(fp1, 0, SEEK_END);
	// size1 = ftell(fp1);
	// rewind(fp1);

	// tc1 = (int*)malloc(sizeof(int) * size1/4);

	// int readnum1 = fread(tc1, 4, size1/4, fp1);

	// fclose(fp1);
	// int judge=0;
	// for(int i=0;i<size1/4;i++)
	// {
	// 	if(tc1[i]-tc2[i] != 0)
	// 	{
	// 		judge++;
	// 	}
	// }
	// printf("judge:%d\n",judge);
	// if (judge == 0) {
	// 	printf("CHALLENGE PASSED\n");
	// }
	// else
	// {
	// 	printf("CHALLENGE FAILED\n");
	// }

	return 0;
}
